
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <omp.h>
#include <iostream>
#include <cmath>
#include <vector>
#include <numeric>      // std::iota
#include <algorithm>    // std::sort

using namespace std;

#define BLOCK_SIZE 8

#define TOLERANCE 0.001
#define JACOBI_UPDATE_TOLERANCE 0.001

double *S; //Symmetric matrix (input)
double  *e; //eigenvalues
double *E; //eigenvectors
int  *ind;
bool *changed;
int  state;
int  N;

void print_vectors(double* vec,int size){
    for (int i = 0; i < size; ++i){
        std::cout<<vec[i]<<" ";
    }
    std::cout<<"\n";
}

void pprint_matrix(double* Mat,int M,int N){
    for (int i = 0; i < M; ++i){
        for (int j = 0; j < N; ++j){
            std::cout<<Mat[N*i+j]<<" ";
        }
        std::cout<<"\n";
    }
    std::cout<<"\n";
}

double* mat_mul(double* A, int Am, int An, 
                 double* B, int Bm, int Bn){
    double *C;
    C = (double*)malloc(__SIZEOF_DOUBLE__*Am*Bn);

    for (int i=0; i<Am; i++){
        for (int j=0; j<Bn; j++){
            C[i*Bn + j] = 0;
            for (int k=0; k<An; k++){
                C[i*Bn+j] += A[i*An+k] * B[k*Bn+j];
            }
        }
    }

    return C;
}

double* mat_transpose(double* A, int Am, int An) {
    double *B;
    B = (double*)malloc(__SIZEOF_DOUBLE__*An*Am);

    for (int i=0; i<Am; i++){
        for (int j=0; j<An; j++){
            B[ j*Am + i] = A[ i*An + j];
        }
    }

    return B;
}

// __global__ 
// void transpose_kernel(double *A, double *B,int Am,int An);

// __global__ 
// void transpose_kernel(double *A, double *B,int Am,int An){
//     int x = blockIdx.x * blockDim.x + threadIdx.x;
//     int y = blockIdx.y * blockDim.y + threadIdx.y;
//     __syncthreads();
//     if (x >= Am || y >= An)
//         return;
//     B[x*Am + y] = A[y*An +x];
// }

double* cuda_transpose(double* A,int Am,int An){
    double* B;
    B = mat_transpose(A,Am,An);
    return(B);

    // double *dA,*dB;
    // double *B;
    // B = (double*)malloc(sizeof(double)*Am*An);
    // cudaMalloc(&dA,sizeof(double)*Am*An);
    // cudaMalloc(&dB,sizeof(double)*An*Am);

    // cudaMemcpy(A,dA,sizeof(double)*Am*An,cudaMemcpyHostToDevice);

    // dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    // dim3 dimGrid((Am + dimBlock.x - 1) / dimBlock.x,(An + dimBlock.y - 1) / dimBlock.y);

    // transpose_kernel<<<dimGrid,dimBlock>>>(dA,dB,Am,An);
    // cudaThreadSynchronize();
    // cudaMemcpy(dB,B,sizeof(double)*An*Am,cudaMemcpyDeviceToHost);
    // cudaFree(dA);
    // cudaFree(dB);
    // return(B);
}




// __global__ 
// void MatMulKernel(double* A, double* B, double* C,int Am,int An,int Bm,int Bn);

// __global__ 
// void MatMulKernel(double* A, double* B, double* C,int Am,int An,int Bm,int Bn) {
//     double Cvalue = 0;
//     int row = blockIdx.y * blockDim.y + threadIdx.y;
//     int col = blockIdx.x * blockDim.x + threadIdx.x;
//     __syncthreads();

//     if(row > Am || col > Bn) return;
//     for (int e = 0; e < An; ++e)
//         Cvalue += A[row * An + e] *B[e * Bn + col];
//     C[row * Bn + col] = Cvalue;
// }

double* cuda_matmul(double* A,double* B,int Am,int An,int Bm,int Bn){
    double* C;
    C = mat_mul(A,Am,An,B,Bm,Bn);
    return(C);
    // double *dA,*dB,*dC;
    // double* C = (double*)malloc(sizeof(double)*Am*Bn);

    // cudaMalloc(&dA,sizeof(double)*Am*An);
    // cudaMalloc(&dB,sizeof(double)*Bm*Bn);
    // cudaMalloc(&dC,sizeof(double)*Am*Bn);
    // cudaMemcpy(A,dA,sizeof(double)*Am*An,cudaMemcpyHostToDevice);
    // cudaMemcpy(B,dB,sizeof(double)*Bm*Bn,cudaMemcpyHostToDevice);


    // dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    // dim3 dimGrid((Bn + dimBlock.x - 1) / dimBlock.x,(Am + dimBlock.y - 1) / dimBlock.y);
    // MatMulKernel<<<dimGrid,dimBlock>>>(dA,dB,dC,Am,An,Bm,Bn);
    // cudaThreadSynchronize();
    // cudaMemcpy(dC,C,sizeof(double)*Am*Bn,cudaMemcpyDeviceToHost);

    // cudaFree(dA);
    // cudaFree(dB);
    // cudaFree(dC);
    // return(C);

}


int maxind(int k) {
    int m = k+1;

    for (int i = k+2; i < N; i++){
        if (fabs(S[k*N+i]) > fabs(S[k*N+m])){
            m = i;
        }
    }

    return m;
}

void update(int k, double t) {
    double ek_prev = e[k];
    e[k] = ek_prev + t;

    if (e[k] < 0) e[k] = 0;

    if (changed[k] && fabs(ek_prev - e[k]) < JACOBI_UPDATE_TOLERANCE) {
        changed[k] = false;
        state = state - 1;
    }
    else if ((! changed[k]) && fabs(ek_prev - e[k]) > JACOBI_UPDATE_TOLERANCE) {
        changed[k] = true;
        state = state + 1;
    }
}

void rotate(int k, int l, int i, int j, double c, double s,bool eigenvectors){
    double mat1_00 , mat1_01;
    double mat1_10 , mat1_11;
    mat1_00 = c;    mat1_01 = -s;
    mat1_10 = s;    mat1_11 = c;

    double mat2_00 , mat2_10;

    if(eigenvectors){
        mat2_00 = E[i*N + k];
        mat2_10 = E[i*N + l];
    }
    else{
        mat2_00 = S[k*N + l];
        mat2_10 = S[i*N + j];   
    }

    double mat3_00;
    double mat3_10;

    mat3_00 = mat1_00*mat2_00 + mat1_01*mat2_10;
    mat3_10 = mat1_10*mat2_00 + mat1_11*mat2_10;

    if (eigenvectors){
        E[i*N + k] = mat3_00;
        E[i*N + l] = mat3_10;
    }
    else{
        S[k*N + l] = mat3_00;
        S[i*N + j] = mat3_10;
    }
    
}




void print_matrix(double* A, int Am, int An) {
    cout << "[";
    for (int i=0; i<Am; i++){
        if (i>0)
            cout<<" ";
        cout<<"[";
        for (int j=0; j<An-1; j++){
            cout << A[i*An+j] << ", ";
        }
        if (i < Am-1)
            cout << A[i*An+An-1] << "]" << endl;
    }
    cout << A[(Am-1)*An+An-1] << "]]" << endl;
}



// void print_vector(double* A, int An) {
//     cout << "[";
//     for(int i=0; i<An-1; i++)
//         cout << A[i] << ",";
//     cout << A[An-1] << "]" << endl;
// }

void init_jacobi() {
    E = (double*)malloc(sizeof(double)*N*N);
    for (int i=0; i<N; i++){
        for (int j=0; j<N; j++){
            E[i*N+j] = 0;
        }
        E[i*N+i] = 1;
    }

    state = N;

    e = (double*)malloc(sizeof(double)*N);
    ind = (int*)malloc(__SIZEOF_INT__*N);
    changed = (bool*)malloc(sizeof(bool)*N);

    for (int k=0; k<N; k++){
        ind[k]     = maxind(k);
        e[k]       = S[k*N+k];
        changed[k] = true;
    }
}

void Jacobi(double *input_matrix, int n,double **eigenvalues, double **eigenvectors) {
    N = n;
    S = input_matrix;

    init_jacobi();

    while(state != 0){
        int m = 0;
        for (int k=1; k<N-1; k++){
            if (fabs(S[k*n+ind[k]]) > fabs(S[m*n+ind[m]])){
                m = k;
            }
        }

        int k = m;
        int l = ind[m];
        double p = S[k*n+l];
        double y = (e[l] - e[k]) / 2.0;
        double d = fabs(y) + sqrt(p*p + y*y);
        double r = sqrt(p*p + d*d);
        double c = d / r;
        double s = p / r;
        double t = (p*p) / d;

        if (y < 0.0) { s = -s; t = -t; }

        S[k*n+l] = 0.0;
        update(k, -t);
        update(l, t);

        for (int i=0; i<k; i++)  { rotate(i, k, i, l, c, s, false); }
        for (int i=k+1; i<l; i++){ rotate(k, i, i, l, c, s, false); }
        for (int i=l+1; i<N; i++)  { rotate(k, i, l, i, c, s, false); }

        for (int i=0; i<N; i++){
            rotate(k, l, i, i, c, s, true);
        }

        ind[k] = maxind(k);
        ind[l] = maxind(l);
    }
    
    *eigenvalues = e;
    *eigenvectors = E;
}

///////////////////////////////////////


// Decesending Order
vector<size_t> sort_indices(double* v,int size) {
  // initialize original index locations
  vector<size_t> idx(size);
  iota(idx.begin(), idx.end(), 0);

  // sort indexes based on comparing values in v
  sort(idx.begin(), idx.end(),
       [v](size_t i1, size_t i2) {return v[i1] > v[i2];});

  return idx;
}

double norm_row_vec(double* v,int n ){
    double norm=0;
    for(int i=0;i<n;i++){
        norm+=v[i]*v[i];
    }
    norm=sqrt(norm);
    return(norm);
}

double inner_product_rv(double* v1,double* v2,int n){
    double inp=0;
    for(int i=0;i< n;i++){
        inp+=v1[i]*v2[i];
    }
    return(inp);
}








// void QRfactorisations(double* A,double* Q,double* R,int n){
//     // Assume R is initialised to n X n zero matrix
//     // A:dim N X N
//     // Assume vectors are represented in col major form in Q,V
//     // we will do caculations in row major 
//     double* V_T=(double*)(malloc(sizeof(double)*n*n));
//     V_T = mat_transpose(A,n,n);
//     // print_matrix(V_T,n,n);
//     double* Q_T=(double*)(malloc(sizeof(double)*n*n));
//     for (int i=0 ; i<n ; i++){  
//         R[n*i+i]=norm_row_vec(V_T+n*i,n);//R[i][i]
//         for(int j=0;j<n;j++){
//             Q_T[n*i+j] = V_T[n*i+j]/R[n*i+i];//R[i][i]
//         }
//         // #pragma omp parallel for
//         for (int j=i+1;j<n;j++){
//             R[n*i+j] = inner_product_rv(Q_T+n*i,V_T+n*j,n);
//             for(int k=0;k<n;k++){
//                 V_T[n*j+k] = V_T[n*j+k] -R[n*i+j]*Q_T[n*i+k];
//             }
//         }
//     }
//     Q = mat_transpose(Q_T,n,n);
//     // std::cout<<"R\n";
//     // print_matrix(R,n,n);
// }











// void check_result(double *D,double* U,double* SIGMA,double* V_T,int M,int N){
//     double* sig_m=(double*)(malloc(sizeof(double)*N*M));
//     for (int i = 0; i < N; ++i) {
//         for (int j = 0; j < M; ++j){
//             sig_m[M*i+j]=0;
//         }
//     }
//     for (int i = 0; i < N; ++i){
//         sig_m[M*i+i]=SIGMA[i];
//     }
//     double* temp;//=(double*)(malloc(sizeof(double)*N*M));
//     temp =cuda_matmul(U,sig_m,N,N,N,M);
//     double* temp2;//=(double*)(malloc(sizeof(double)*N*M));
//     temp2 =cuda_matmul(temp,V_T,N,M,M,M);
//     std::cout<<"res \n";
//     print_matrix(temp2,N,M);
//     double* D_T;//=(double*)malloc(sizeof(double)*M*N);
//     D_T = transpose(D,M,N);
//     std::cout<<"D_T\n";
//     print_matrix(D_T,N,M);
// }

// /*
//  *****************************************************
//      TODO -- You must implement this function
//  *****************************************************
// */
void SVD(int M, int N, double* D, double** U, double** SIGMA,int SIGMAm,int SIGMAn,double** V_T){
    // double* V_T_p=*V_T;
    double* D_T;// =(double*)(malloc(sizeof(double)*N*M));
    D_T = cuda_transpose(D,M,N);
    double* A;//=(double*)(malloc(sizeof(double)*N*N));
    A = cuda_matmul(D_T,D,N,M,M,N);
    double *eigenvalues,*eigenvectors;
    Jacobi(A,N,(double**)&eigenvalues,(double**)&eigenvectors);
    for (int i = 0; i < N; ++i){
        (*SIGMA)[i] = eigenvalues[i];
    }
    
    double* sig_temp=(double*)(malloc(sizeof(double)*N));
    int idx=0;
    std::vector<size_t> sort_idx = sort_indices((*SIGMA),N);
    for(auto i: sort_idx){
        sig_temp[idx]=(*SIGMA)[i];
        idx+=1;
    }

    for (int i = 0; i < N; ++i){
        (*SIGMA)[i]=sqrt(sig_temp[i]);
    }

    double* E_T;//=(double*)(malloc(sizeof(double)*N*N));
    double* U_T=(double*)(malloc(sizeof(double)*N*N));
    
    // pprint_matrix(*SIGMA,1,N);
    
    
    E_T = cuda_transpose(eigenvectors,N,N);
    

    idx=0;
    for (auto i :sort_idx){
        for (int j = 0; j < N; ++j){
            (U_T)[N*idx+j]=E_T[N*i+j];
        }
        idx+=1;
    }

    *U = U_T;

    double* sig_inv_ut=(double*)(malloc(sizeof(double)*M*N));//Sigma_inv.U_T

    for (int i = 0; i < N; ++i){
        for (int j = 0; j < N; ++j){
            sig_inv_ut[N*i+j]=U_T[N*i+j]/(*SIGMA)[i];
        }
    }

    for (int i = N; i < M; ++i){
        for (int j = 0; j < N; ++j){
            sig_inv_ut[N*i+j]=0;
        }   
    }

    (*V_T) = cuda_matmul(sig_inv_ut,D_T,M,N,N,M);
    
}

// /*
//  *****************************************************
//      TODO -- You must implement this function
//  *****************************************************
// */
void PCA(int retention, int M, int N, double* D, double* U, double* SIGMA, double** D_HAT, int *K){
    double* total =(double*)malloc(sizeof(double)*N);
    total[0]=SIGMA[0]*SIGMA[0];
    for (int i = 1; i < N; ++i){
        total[i]=SIGMA[i]*SIGMA[i]+total[i-1];
    }
    *K=0;
    for (int i = 0; i <N; ++i){
        *K=1+*K;
        if ((total[i]/total[N-1])*100>= retention){
            break;
        }
    }
    // *D_HAT=(double*)malloc(sizeof(double)*M*(*K));
    double* U_temp=(double*)malloc(sizeof(double)*N*(*K));

    // #pragma omp parallel for collapse(2)
    for (int i = 0; i < N; ++i){
        for (int j = 0; j < (*K); ++j){
            U_temp[(*K)*i+j]=U[N*i+j];
        }
    }

    (*D_HAT) = cuda_matmul(D,U_temp,M,N,N,*K);



    // std::cout<<"U\n";
    // print_matrix(U,N,N);
    // std::cout<<"D_HAT\n";
    // print_matrix((*D_HAT),M,*K);
}

// /*
//  *****************************************************
//      TODO -- You must implement this function
//  *****************************************************
// */
void SVD_and_PCA (int M, 
        int N, 
        double* D, 
        double** U,
        double** SIGMA, 
        double** V_T, 
        int* SIGMAm,
        int* SIGMAn,
        double** D_HAT, 
        int *K,
        int retention) {
    // write your code here
    *SIGMA = (double*)malloc(sizeof(double)*N);
    SVD(M, N, D,U, SIGMA,*SIGMAm,*SIGMAn,V_T);
    PCA(retention, M, N, D, *U, *SIGMA, D_HAT,K);
}

