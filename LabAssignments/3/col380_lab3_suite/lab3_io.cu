#include "hip/hip_runtime.h"
#include "lab3_io.h"
#include "Eigen/Eigen/Dense"

#include <utility>

using namespace Eigen;
void read_matrix (const char* input_filename, int* M, int* N, double** D){
	FILE *fin = fopen(input_filename, "r");
	int i;

	fscanf(fin, "%d%d", M, N);
	
	int num_elements = (*M) * (*N);
	*D = (double*) malloc(sizeof(double)*(num_elements));
	
	for (i = 0; i < num_elements; i++){
		fscanf(fin, "%lf", (*D + i));
	}
	fclose(fin);
}

MatrixXf ptr_to_mat(int M, int N, double *ptr)
{
  MatrixXf ret(M,N);
  for (size_t i = 0; i < M; ++i)
    for (size_t j = 0; j < N; ++j)
      ret(i,j) = *(ptr+i*N+j);
  return ret;
}

std::pair<bool, double> compare_under_tolerance(MatrixXf const& m, MatrixXf const& n, double tolerances[], size_t t_sz)
{
  assert(m.rows() == n.rows());
  assert(m.cols() == n.cols());
  double m_norm = m.norm();
  double n_norm = n.norm();

  double m_norm_abs = fabs(m_norm);
  double n_norm_abs = fabs(n_norm);
  double d = fabs(m_norm_abs-n_norm_abs)*100.0f/n_norm_abs;
  for (size_t i = 0; i < t_sz; ++i) {
    if (d <= tolerances[i]) {
      return std::make_pair(true, tolerances[i]);
    }
  }
  return std::make_pair(false, 100.0f);
}

void write_result (int M, 
		int N, 
		double* D, 
		double* U, 
		double* SIGMA, 
		double* V_T,
		int SIGMAm, 
		int SIGMAn, 
		int K, 
		double* D_HAT,
		double computation_time){
	// Will contain output code
	// printf("hello Cuda\n");
	double tolerances[] = { 0.001, 0.01, 0.1, 1.0 };

  MatrixXf Dm = ptr_to_mat(M, N, D);
  MatrixXf Um = ptr_to_mat(N, N, U);
  MatrixXf Vm = ptr_to_mat(M, M, V_T);
  MatrixXf sigma = MatrixXf::Zero(N,M);
  for (size_t i = 0; i < N; ++i)
    sigma(i, i) = *(SIGMA+i);

  MatrixXf Dm_n = Um*sigma*Vm;
  Dm_n.transposeInPlace();

  auto p = compare_under_tolerance(Dm, Dm_n, tolerances, sizeof(tolerances)/sizeof(double));
  bool d_equal = p.first;
  double d_tolerance = p.second;


  printf("Time: %lf\n", computation_time);
#define FLAG(f) ((f) ? 'T' : 'F')
  printf("%c %lf\n", FLAG(d_equal), d_tolerance);
}
